#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <math.h>

#define N 6
#define UPPER 1
#define LOWER N*3
#define THREADS_PER_BLOCK 1
#define BLOCKS 1

void rand_init_array(int *a, int n, int upper, int lower);
void display_array(int *a, int n);

__global__ void setup_kernel(hiprandState *state){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(1234, idx, 0, &state[idx]);
}

__device__ int random(hiprandState *state, unsigned int n){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    float myrandf = hiprand_uniform(&(state[idx]));
    myrandf *= ((n-1) + 0.999999);
    int myrand = (int)truncf(myrandf);
    return myrand;
}

/*
 * Function:  swap_random 
 * --------------------
 * Randomizes elements of array
 *
 *  a: the array (integer)
 *  i: the index of element that will be swapped
 *  n: number of elements in the array
 *
 */

 __device__ void swap_random(int *a, int idx, int n, hiprandState *state){
    int t1, t2, r;
    do
        r = random(state, n%idx);
    while(r == idx);
    t1 = a[idx];
    t2 = a[r];
    atomicExch(&(a[idx]), t2);
    atomicExch(&(a[r]), t1);
    // a[idx] = a[r];
    // a[r] = t;
    // printf("idx: %d, a[%d]: %d | r: %d, a[%d]: %d\n", idx, idx, a[idx], r, r, a[r]);
    __threadfence();
}

/*
 * Function:  shuffle 
 * --------------------
 * Randomizes elements of array
 *
 *  a: the array (integer)
 *  n: number of elements in the array
 *
 */

__device__ void shuffle(int *a, int n, hiprandState *state){
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    swap_random(a, index, n, state);
}

/*
 * Function:  is_sorted 
 * --------------------
 * Checks if array is sorted
 *
 *  a: the array (integer)
 *  n: number of elements in the array
 *
 */

__device__ int is_sorted(int *a, int n){
    while ( --n >= 1 )
        if ( a[n] < a[n-1] ) return 0;
    return 1;
}

/*
 * Function:  bogo_sort 
 * --------------------
 * Performs bogo sort (random suffle until the array is sorted)
 *
 *  a: the array (integer)
 *  n: number of elements in the array
 *
 */
 __global__ void bogo_sort(int *a, int n, volatile int *found, hiprandState *state){
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < n){
        while(!found[0]){
            shuffle(a, n, state);
            found[0] = is_sorted(a, n);
        }
    }
}

/*
 * Main
 */
 int main(int argc, char *argv[]){
    
    float total_time, comp_time;
    hipEvent_t total_start, total_stop, comp_start, comp_stop;
    hipEventCreate(&total_start);
  	hipEventCreate(&total_stop);
  	hipEventCreate(&comp_start);
    hipEventCreate(&comp_stop);

    /*
	 * Memory allocation on host 
	 */
    int *array = (int *)malloc(N*sizeof(int));
    int *found = {0};

    /*
     * Init array
     */
    rand_init_array(array, N, UPPER, LOWER);
    display_array(array, N);
 
    /*
     * Memory allocation on device
     */
    int *array_dev, *found_dev;
    hipMalloc((void **)&array_dev, N*sizeof(int));
    hipMalloc((void **)&found_dev, 1*sizeof(int));
     
    hipEventRecord(total_start);
 
    /*
     * Copy array from host memory to device memory
     */
    hipMemcpy(array_dev, array, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(found_dev, found, N*sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(comp_start);

    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));

    setup_kernel<<< BLOCKS, THREADS_PER_BLOCK >>>(d_state);

    /*
     * Kernel call
     */ 
    bogo_sort<<< BLOCKS, THREADS_PER_BLOCK >>>(array_dev, N, found_dev, d_state);
 
    hipEventRecord(comp_stop);
    hipEventSynchronize(comp_stop);
    hipEventElapsedTime(&comp_time, comp_start, comp_stop);
 
    /*
     * Copy c from host device memory to host memory
     */
    hipMemcpy(array, array_dev, N*sizeof(int), hipMemcpyDeviceToHost);
    
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_time, total_start, total_stop);

    /*
     * Free memory on device
     */
    hipFree(array_dev);
    hipEventDestroy(comp_start);
    hipEventDestroy(comp_stop);
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);
        
    /*
     * GPU timing
     */
    printf("N: %d, blocks: %d, total_threads: %d\n", N, BLOCKS, THREADS_PER_BLOCK*BLOCKS);
    printf("Total time (ms): %f\n", total_time);
    printf("Kernel time (ms): %f\n", comp_time);
    printf("Data transfer time (ms): %f\n", total_time-comp_time);
    display_array(array, N);
         
    return 0;
      
 }

/*
 * Function:  rand_init_array 
 * --------------------
 * Fills an integer array with random numbers
 *
 *  a: the array that will be filled with numbers
 *  n: number of elements in the array
 *  upper: highest value of random number
 *  lower: lowest value of random number
 *
 */

void rand_init_array(int *a, int n, int upper, int lower){
    int i;    
    for (i=0; i<n; ++i)
        a[i] = (rand() % (upper - lower + 1)) + lower;
}

/*
 * Function:  display_array 
 * --------------------
 * Prints an integer array to user
 *
 *  a: the array that will be printed
 *  n: number of elements in the array
 *
 */

void display_array(int *a, int n){
    int i;
    for (i=0; i < n; ++i) printf("%d ", a[i]);
    printf("\n");
}