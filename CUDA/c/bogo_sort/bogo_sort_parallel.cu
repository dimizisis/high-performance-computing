#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define N 6
#define UPPER 1
#define LOWER N*4
#define THREADS_PER_BLOCK 1
#define BLOCKS 1 //(N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK

void rand_init_array(int *a, int n, int upper, int lower);
void display_array(int *a, int n);

__global__ void setup_kernel(hiprandState *state){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(clock64(), idx, 0, &state[idx]);
}

__device__ float getnextrand(hiprandState *state){

    return (float)(hiprand_uniform(state));
}
  
__device__ int getnextrandscaled(hiprandState *state, int scale){
  
    return (int) scale * getnextrand(state);
}

/*
 * Function:  swap_random 
 * --------------------
 * Randomizes elements of array
 *
 *  a: the array (integer)
 *  i: the index of element that will be swapped
 *  n: number of elements in the array
 *
 */

__device__ void swap_random(int *a, hiprandState *state){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int r = getnextrandscaled(state, N);
    a[r] = atomicExch(&(a[idx]), a[r]);
    printf("%d ", r);

}

/*
 * Function:  is_sorted 
 * --------------------
 * Checks if array is sorted
 *
 *  a: the array (integer)
 *  n: number of elements in the array
 *
 */

__device__ int is_sorted(int *a, int n){
    while ( --n >= 1 )
        if ( a[n] < a[n-1] || a[n] == a[n-1]) return 0;
    return 1;
}

/*
 * Function:  bogo_sort 
 * --------------------
 * Performs bogo sort (random suffle until the array is sorted)
 *
 *  a: the array (integer)
 *  n: number of elements in the array
 *
 */

__global__ void bogo_sort(int *a, int n, int *found, hiprandState *state){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n){
        while(!found[0]){
            swap_random(a, state);
            found[0] = is_sorted(a, n);
        }
    }
}

/*
 * Main
 */

int main(int argc, char *argv[]){
    
    float total_time, comp_time;
    hipEvent_t total_start, total_stop, comp_start, comp_stop;
    hipEventCreate(&total_start);
  	hipEventCreate(&total_stop);
  	hipEventCreate(&comp_start);
    hipEventCreate(&comp_stop);

    /* Some initializations & allocations to generate random number within kernel */

    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));
    
    setup_kernel<<< BLOCKS, THREADS_PER_BLOCK >>>(d_state);
    /* -------------------------------------------------------------------------- */

    /*
	 * Memory allocation on host 
	 */
    int *array = (int *)malloc(N*sizeof(int));
    int *found = {0};

    /*
     * Init array
     */
    rand_init_array(array, N, UPPER, LOWER);
    display_array(array, N);
 
    /*
     * Memory allocation on device
     */
    int *array_dev, *found_dev;
    hipMalloc((void **)&array_dev, N*sizeof(int));
    hipMalloc((void **)&found_dev, 1*sizeof(int));
     
    hipEventRecord(total_start);
 
    /*
     * Copy array from host memory to device memory
     */
    hipMemcpy(array_dev, array, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(found_dev, found, 1*sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(comp_start);

    /*
     * Kernel call
     */ 
    bogo_sort<<< BLOCKS, THREADS_PER_BLOCK >>>(array_dev, N, found_dev, d_state);
 
    hipEventRecord(comp_stop);
    hipEventSynchronize(comp_stop);
    hipEventElapsedTime(&comp_time, comp_start, comp_stop);
 
    /*
     * Copy c from host device memory to host memory
     */
    hipMemcpy(array, array_dev, N*sizeof(int), hipMemcpyDeviceToHost);
    
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_time, total_start, total_stop);

    /*
     * Free memory on device
     */
    hipFree(array_dev);
    hipEventDestroy(comp_start);
    hipEventDestroy(comp_stop);
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);
        
    /*
     * GPU timing
     */
    printf("N: %d, blocks: %d, total_threads: %d\n", N, BLOCKS, THREADS_PER_BLOCK*BLOCKS);
    printf("Total time (ms): %f\n", total_time);
    printf("Kernel time (ms): %f\n", comp_time);
    printf("Data transfer time (ms): %f\n", total_time-comp_time);
    display_array(array, N);
         
    return 0;
      
 }

/*
 * Function:  rand_init_array 
 * --------------------
 * Fills an integer array with random numbers
 *
 *  a: the array that will be filled with numbers
 *  n: number of elements in the array
 *  upper: highest value of random number
 *  lower: lowest value of random number
 *
 */

void rand_init_array(int *a, int n, int upper, int lower){
    int i;    
    for (i=0; i<n; ++i)
        a[i] = (rand() % (upper - lower + 1)) + lower;
}

/*
 * Function:  display_array 
 * --------------------
 * Prints an integer array to user
 *
 *  a: the array that will be printed
 *  n: number of elements in the array
 *
 */

void display_array(int *a, int n){
    int i;
    for (i=0; i < n; ++i) printf("%d ", a[i]);
    printf("\n");
}