#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 128
#define base 0
#define THREADS_PER_BLOCK 512

__global__ void count_characters(char *buffer, int *freq, long file_size, int total_threads);

void display_count(int *freq, int n);

/*
 * Main
 */

int main(int argc, char *argv[]){	
    int blocks;	
    int num_threads;
     
    float total_time, comp_time;
    hipEvent_t total_start, total_stop, comp_start, comp_stop;
    hipEventCreate(&total_start);
  	hipEventCreate(&total_stop);
  	hipEventCreate(&comp_start);
    hipEventCreate(&comp_stop);
      
    FILE *pFile;
	long file_size;
	char * buffer;
	char * filename;
	size_t result;
	int * freq;

    if (argc != 2) {
		printf ("Usage : %s <file_name>\n", argv[0]);
		return 1;
    }

	filename = argv[1];
	pFile = fopen ( filename , "rb" );
	if (pFile==NULL) {printf ("File error\n"); return 2;}

	/* obtain file size */
	fseek (pFile , 0 , SEEK_END);
	file_size = ftell (pFile);
	rewind (pFile);
	printf("file size is %ld\n", file_size);
	
	/* allocate memory to contain the file	*/
	buffer = (char*) malloc (sizeof(char)*file_size);
	if (buffer == NULL) {printf ("Memory error\n"); return 3;}

	/* copy the file into the buffer */
	result = fread (buffer,1,file_size,pFile);
    if (result != file_size) {printf ("Reading error\n"); return 4;} 
    
    freq = (int*) malloc(sizeof(int)*N);
    if (freq == NULL) {printf ("Memory error\n"); return 5;}

	/*
	 * Memory allocation on device
	 */
    char *buff_dev;
    int *freq_dev;
    hipMalloc((void **)&buff_dev, file_size*sizeof(char));
    hipMalloc((void **)&freq_dev, N*sizeof(int));
    hipMemset(freq_dev, 0, N);
	
    hipEventRecord(total_start);

    /*
	 * Copy buffer from host memory to device memory
	 */
	hipMemcpy(buff_dev, buffer, sizeof(char)*file_size, hipMemcpyHostToDevice);
	
    /*
    * Create sufficient blocks 
    */
    blocks = (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    /*
    * Calculate number of threads
    */
    num_threads = blocks * THREADS_PER_BLOCK;

    hipEventRecord(comp_start);
	/*
    * Kernel call
    */ 
	count_characters<<< blocks*2, N >>>(buff_dev, freq_dev, file_size, num_threads);

    hipEventRecord(comp_stop);
    hipEventSynchronize(comp_stop);
    hipEventElapsedTime(&comp_time, comp_start, comp_stop);

	/*
	 * Copy c from host device memory to host memory
	 */
	hipMemcpy(freq, freq_dev, N*sizeof(int), hipMemcpyDeviceToHost);
	
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_time, total_start, total_stop);
	/*
	 * Free memory on device
     */
    hipFree(buff_dev);
    hipFree(freq_dev);
    hipEventDestroy(comp_start);
    hipEventDestroy(comp_stop);
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);

    /*
    * Display Results
    */
    display_count(freq, N);
       
    /*
    * GPU timing
    */
    printf("N: %d, blocks: %d, total_threads: %d\n", N, blocks, THREADS_PER_BLOCK*blocks);
    printf("Total time (ms): %f\n", total_time);
    printf("Kernel time (ms): %f\n", comp_time);
    printf("Data transfer time (ms): %f\n", total_time-comp_time);    
        
	return 0;
}

/*
 * Function:  count_characters 
 * --------------------
 * Counts the frequency of each character (atomic operation, freq array)
 *
 *  buffer: pointer to char array that contains the txt file
 *  freq: pointer to int array that will contain the frequency of each character
 *  file_size: the size of the file (long number)
 *  total_threads: calculated total threads (int)
 *
 */

__global__ void count_characters(char *buffer, int *freq, long file_size, int total_threads){
	
    int index = threadIdx.x + blockIdx.x * blockDim.x;     
    
    long i;
    for (i=index; i<file_size; i+=total_threads)
        atomicAdd(&(freq[buffer[i] - base]), 1);
        
}

void display_count(int *freq, int n){
	int j;
	for (j=0; j<n; ++j)
		(void) printf("%d = %d\n", j+base, freq[j]);
}