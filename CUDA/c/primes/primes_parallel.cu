
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 20
#define THREADS_PER_BLOCK 512
#define BLOCKS (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK

__global__ void find_primes(int *a, int n) { 

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // int total_threads = gridDim.x * blockDim.x;
    int is_prime = 1;

    if (idx > 1 && idx < n){
        int j;
        for (j=2; j<idx/2+1; ++j){
            if (!(idx % j) && j != idx){
                is_prime = 0;
                break;
            }
        }
        if (is_prime) a[idx] = 1;
        is_prime = 1;
    }

}

int main(int argc, char *argv[]) {
  
    float total_time, comp_time;
    hipEvent_t total_start, total_stop, comp_start, comp_stop;
    hipEventCreate(&total_start);
  	hipEventCreate(&total_stop);
  	hipEventCreate(&comp_start);
  	hipEventCreate(&comp_stop);

    /*
    * Host's array
    */
    int *array;
    array = (int*) calloc(N, sizeof(int));

    /*
    * Start counting total time
    */
    hipEventRecord(total_start);

    /*
    * Device's array
    */
    int *dev_array;
    hipMalloc(&dev_array, N * sizeof(int));

    /*
    * Start counting compile time
    */
    hipEventRecord(comp_start);

    /*
    * Kernel call
    */
    find_primes<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_array, N);

    /*
    * Compile time count
    */

    hipEventRecord(comp_stop);
    hipEventSynchronize(comp_stop);
    hipEventElapsedTime(&comp_time, comp_start, comp_stop);

    /*
	 * Copy c from host device memory to host memory
	 */
    hipMemcpy(array, dev_array, N*sizeof(int), hipMemcpyDeviceToHost);
    
    /*
    * Total time count
    */        
    hipEventRecord(total_stop);
    hipEventSynchronize(total_stop);
    hipEventElapsedTime(&total_time, total_start, total_stop);

	/*
	 * Free memory on device
	 */
    hipFree(dev_array);
    hipEventDestroy(comp_start);
    hipEventDestroy(comp_stop);
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);

    /*
    * GPU timing
    */
    printf("N: %d, blocks: %d, total_threads: %d\n", N, BLOCKS, THREADS_PER_BLOCK*BLOCKS);
    printf("Total time (ms): %f\n", total_time);
    printf("Kernel time (ms): %f\n", comp_time);
    printf("Data transfer time (ms): %f\n", total_time-comp_time);

    /*
    * Printing primes
    */
    (void) printf("\n\nPrimes: [ ");
    int i;
    for (i=2; i<=N; ++i)
        if (array[i])
            (void) printf("%d ", i);
    (void) printf("]\n\n");

    return 0;
}